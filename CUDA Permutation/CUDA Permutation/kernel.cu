#include "hip/hip_runtime.h"
#pragma once
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <cstdio>
#include <ctime>
#include <cmath>
#include <string>
#include<iostream>

#include "InputCSV.h"
#include "OutputCSV.h"
// reel set define
#define REEL_WILD -1
// winning set define
#define WINNING_NONE 0
#define WINNING_ANY  -1

using namespace std;

__device__ bool Compare(int* set, int* winningSet, int size) {
	// init any.
    int Any = WINNING_ANY;
    
	for (int i = 0; i < size; i++) {
		if(set[i] != REEL_WILD){
			if(winningSet[i] == WINNING_ANY){
				// first set any
				if(Any == WINNING_ANY){
					Any = set[i];
				} else{
					if(set[i] != Any){
						return false;
					}
				}
			}else if(winningSet[i] != WINNING_NONE){
				 // ordinary compare
				 if(set[i] != winningSet[i]){
					 return false;
				 }
			 }
		}   
    }
    return true;
}

// 設定每個kernel的亂數種子
__global__ void SetupCurand(hiprandState *state, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}
// 跑模擬
__global__ void Simulate(hiprandState *states, const int colunmSize, const int rowSize, int* reelSets, const int reelSetSize, int* payTable, int winningSetSize, size_t runTimes, size_t* hitTimes, size_t* noHitTimes, const size_t NUM_OF_THREAD) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState localState = states[idx];
    int* set;
    set = (int*) malloc(colunmSize*rowSize * sizeof(int));
    for (int t = 0; t < runTimes; t++) {
        for (int col = 0; col < colunmSize; col++) {
            unsigned int rand = hiprand(&localState) % reelSetSize;
            for (int row = 0; row < rowSize; row++) {
                set[row + col * rowSize] = reelSets[(rand + row) % reelSetSize];
            }
        }
        bool hit = false;
        for (int n = 0; n < winningSetSize; n++) {
            if (Compare(set, (payTable + colunmSize * rowSize * n), colunmSize * rowSize)) {
                hit = true;
                // Hit.
                hitTimes[idx + n * NUM_OF_THREAD] += 1;
            }
        }
        // 紀錄no hit.
        if (!hit) { noHitTimes[idx] += 1; }
        states[idx] = localState;
    }
    free(set);
};

int main(int argc, char** argv) {
    const unsigned int RUN_TIMES = 50000000;

    // 加入 Console 參數
    if (argc != 3) { printf(".exe [input file] [output file]\n"); return 1; }
    string intputPath = argv[1];
    string outputPath = argv[2];

    // 設定輸入輸出檔案
    InputCSV inputFile(intputPath);
    OutputCSV outputFile(outputPath);

    // 計時開始
    unsigned long cStart = clock();

    // 模擬 Column*Row 的盤面.
    const int COLUMN_SIZE = inputFile.getPermutationColumnSize();
    const int REEL_ROW_SIZE = inputFile.getReelRowSize();

    // Symbols
    const string *SYMBOLS = inputFile.getPermutationElements();
    const int SYMBOLS_SIZE = inputFile.getPermutationElementsCount();

    // Stops.(停止點 = Reel上的元素)
    const int* STOPS = inputFile.getReelSet();
    const int STOPS_SIZE = inputFile.getReelSetSize();

    // Pay Table
    const int* PAY_TABLE = inputFile.getPayTable();
    // Size of pay table(Element count).
    const int PAY_TABLE_SIZE = inputFile.getPayTableSize();

    // PAY_TABLE 裡變數的數量 = sizeof(PAY_TABLE) / sizeof(int)
    const int PAY_TABLE_REAL_SIZE = PAY_TABLE_SIZE * COLUMN_SIZE * REEL_ROW_SIZE;

    //---------------------Begin of cuda-----------------------------
    size_t *hitTimes;
    size_t *host_hitTimes;
    size_t *dev_hitTimes;

    size_t *host_noHitTimes;
    size_t *dev_noHitTimes;


    int* dev_reelSets;
    int* dev_winningSets;


    // 設定 thread & block.
    size_t threads = 100;
    size_t blocks = 1000;

    size_t NumOfThread = blocks * threads, kernelRunTimes = ceil(RUN_TIMES / NumOfThread);
    printf("Total times: %d\nBlock count: %d\nThread count: %d\nKernelRunTimes: %d\n", RUN_TIMES, blocks, threads, kernelRunTimes);

    // 配置Host memory.
    hitTimes = (size_t*) malloc(PAY_TABLE_SIZE * sizeof(size_t));
    host_hitTimes = (size_t*) malloc(NumOfThread * PAY_TABLE_SIZE * sizeof(size_t));
    host_noHitTimes = (size_t*) malloc(NumOfThread * sizeof(size_t));


    // 配置Device memory.
    hipMalloc((void**) &dev_hitTimes, NumOfThread * PAY_TABLE_SIZE * sizeof(size_t));
    hipMalloc((void**) &dev_noHitTimes, NumOfThread * sizeof(size_t));


    // Declare reel sets.
    hipMalloc((void**) &dev_reelSets, STOPS_SIZE * sizeof(int));
    hipMemcpy(dev_reelSets, STOPS, STOPS_SIZE * sizeof(int), hipMemcpyHostToDevice);

    // Declare pay table.
    hipMalloc((void**) &dev_winningSets, PAY_TABLE_REAL_SIZE * sizeof(int));
    hipMemcpy(dev_winningSets, PAY_TABLE, PAY_TABLE_REAL_SIZE * sizeof(int), hipMemcpyHostToDevice);

    // Setup random seed for each threads.
    hiprandState* devStates;
    hipMalloc(&devStates, NumOfThread * sizeof(hiprandState));
    SetupCurand <<<blocks, threads >>> (devStates, time(NULL));

    // Simulate.
    Simulate <<<blocks, threads >>> (devStates, COLUMN_SIZE, REEL_ROW_SIZE, dev_reelSets, STOPS_SIZE, dev_winningSets, PAY_TABLE_SIZE, kernelRunTimes, dev_hitTimes, dev_noHitTimes, NumOfThread);

    // Copy device memory to host.
    hipMemcpy(host_hitTimes, dev_hitTimes, NumOfThread * PAY_TABLE_SIZE * sizeof(size_t), hipMemcpyDeviceToHost);
    hipMemcpy(host_noHitTimes, dev_noHitTimes, NumOfThread * sizeof(size_t), hipMemcpyDeviceToHost);


    //釋放Memory.
    hipFree(dev_reelSets);
    hipFree(dev_winningSets);
    hipFree(dev_hitTimes);

    //---------------------End of cuda-----------------------------
    // 算 Hit.
    size_t totalHitTimes = 0;
    for (size_t i = 0; i < PAY_TABLE_SIZE; i++) {
        hitTimes[i] = 0;
        for (size_t t = 0; t < NumOfThread; t++) {
            hitTimes[i] += host_hitTimes[i * NumOfThread + t];
        }
        totalHitTimes += hitTimes[i];
    }
    // 算 No Hit.
    size_t noHitTimes = 0;
    for (size_t t = 0; t < NumOfThread; t++) {
        noHitTimes += host_noHitTimes[t];
    }
    // 計時完了
    unsigned long spendTime = clock() - cStart;

    // Console print.
    printf("CUDA run %lu ms.\n", spendTime);
    printf("Output to %s... \n", outputPath.c_str());

    // 輸出
    outputFile.WriteTitle(blocks, threads, RUN_TIMES, spendTime, STOPS_SIZE, COLUMN_SIZE, REEL_ROW_SIZE, totalHitTimes, (double)totalHitTimes / RUN_TIMES);
    
    // Output No hit frequency.
    outputFile.WriteHitFreq("No Hit", noHitTimes, (double) noHitTimes / RUN_TIMES);

    // Output hit frequency.
    for (int i = 0; i < PAY_TABLE_SIZE; i++) {
        outputFile.WriteHitFreq(inputFile.getPayTableFileName(i), hitTimes[i], (double) hitTimes[i] / RUN_TIMES);
    }



    outputFile.Close();
    delete[] hitTimes;
    printf("Finish.\n");
    return 0;
}
